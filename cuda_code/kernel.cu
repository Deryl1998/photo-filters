#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
extern "C" __declspec(dllexport) int main(unsigned char* oldData, int width, int height, int delta,int choise);

__device__ int getValue(int value, int b)
{
    if (value + b < 0) return 0;
    if (value + b > 255) return 255;
    return value + b;
}

__global__ void brightUp(unsigned char* oldData, int width, int height, int delta)
{
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = blockId * blockDim.x + threadIdx.x;
    if (threadId >= height*width*4 || threadId%3==0) return;
    int colorPixel = (int)(oldData[threadId]);
    oldData[threadId] = getValue(colorPixel, delta);
}


__global__ void contrastUp(unsigned char* oldData, int width, int height, int contrast)
{
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = blockId * blockDim.x + threadIdx.x;
    if (threadId >= height * width * 4 || threadId % 3 == 0) return;
     int colorPixel = static_cast<int>(oldData[threadId]);
     colorPixel = getValue(colorPixel, -128);
     colorPixel *= contrast;
     colorPixel = getValue(colorPixel, 128);
     oldData[threadId] = colorPixel;
}

__global__ void thresholdingUp(unsigned char* oldData, int width, int height, int strength)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = blockId * blockDim.x + threadIdx.x;
    if (threadId >= height * width * 4 || threadId % 3 == 0) return;
   
    int gray = oldData[threadId + 0] * (0.299 + (0.03 * strength)) +
        oldData[threadId + 1] * (0.587 + (0.05 * strength)) +
        oldData[threadId + 2] * (0.114 + (0.01 * strength));
    oldData[threadId + 0] = oldData[threadId + 1];
    oldData[threadId + 1] = oldData[threadId + 2];
    oldData[threadId + 2] = gray;

}

int main(unsigned char* oldData, int width, int height, int power, int choise)
{
    const int size = width * height * 4;
    unsigned char* dev_bitmap = new unsigned char[size];
    dim3 grid(width, height);
    hipMalloc(&dev_bitmap, size * sizeof(unsigned char));
    hipMemcpy(dev_bitmap, oldData, size * sizeof(unsigned char), hipMemcpyHostToDevice);

    switch (choise) {
    case 0:  brightUp << < grid, 4 >> > (dev_bitmap, width, height, power); break;
    case 1:  contrastUp << < grid, 4 >> > (dev_bitmap, width, height, power); break;
    case 2:  thresholdingUp << < grid, 4 >> > (dev_bitmap, width, height, power); break;
    }
    hipDeviceSynchronize();
    hipMemcpy(oldData, dev_bitmap, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipFree(dev_bitmap);
    return 0;
}